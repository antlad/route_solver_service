#include "hip/hip_runtime.h"
#include "cuda_func_opt.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <Eigen/LU>
#include <Eigen/Core>
#include <Eigen/Dense>

#include <iostream>

int ei_test_init_cuda()
{
	int device = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);
	std::cout << "CUDA device info:\n";
	std::cout << "  name:                        " << deviceProp.name << "\n";
	std::cout << "  capability:                  " << deviceProp.major << "." << deviceProp.minor << "\n";
	std::cout << "  multiProcessorCount:         " << deviceProp.multiProcessorCount << "\n";
	std::cout << "  maxThreadsPerMultiProcessor: " << deviceProp.maxThreadsPerMultiProcessor << "\n";
	std::cout << "  warpSize:                    " << deviceProp.warpSize << "\n";
	std::cout << "  regsPerBlock:                " << deviceProp.regsPerBlock << "\n";
	std::cout << "  concurrentKernels:           " << deviceProp.concurrentKernels << "\n";
	std::cout << "  clockRate:                   " << deviceProp.clockRate << "\n";
	std::cout << "  canMapHostMemory:            " << deviceProp.canMapHostMemory << "\n";
	std::cout << "  computeMode:                 " << deviceProp.computeMode << "\n";
	return 1;
}


static void HandleError( hipError_t err, const char *file, int line )
{
	// CUDA error handeling from the "CUDA by example" book
	if (err != hipSuccess)
	{
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


	__global__ void cu_dot(Eigen::Vector3d *v1, Eigen::Vector3d *v2, double *out, size_t N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if(idx < N)
		{
			out[idx] = v1[idx].dot(v2[idx]);
		}
		return;
	}

	// The wrapper for the calling of the actual kernel
	double dot_cuda(const std::vector<Eigen::Vector3d> & v1, const std::vector<Eigen::Vector3d> & v2)
	{
		int n = v1.size();
		double *ret = new double[n];

		// Allocate device arrays
		Eigen::Vector3d *dev_v1, *dev_v2;
		HANDLE_ERROR(hipMalloc((void **)&dev_v1, sizeof(Eigen::Vector3d)*n));
		HANDLE_ERROR(hipMalloc((void **)&dev_v2, sizeof(Eigen::Vector3d)*n));
		double* dev_ret;
		HANDLE_ERROR(hipMalloc((void **)&dev_ret, sizeof(double)*n));

		// Copy to device
		HANDLE_ERROR(hipMemcpy(dev_v1, v1.data(), sizeof(Eigen::Vector3d)*n, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(dev_v2, v2.data(), sizeof(Eigen::Vector3d)*n, hipMemcpyHostToDevice));

		// Dot product
		cu_dot<<<(n+1023)/1024, 1024>>>(dev_v1, dev_v2, dev_ret, n);

		// Copy to host
		HANDLE_ERROR(hipMemcpy(ret, dev_ret, sizeof(double)*n, hipMemcpyDeviceToHost));

		// Reduction of the array
		for (int i=1; i<n; ++i)
		{
			ret[0] += ret[i];
		}

		// Return
		return ret[0];
	}

